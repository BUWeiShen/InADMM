#include "mex.h"
#include "gpu/mxGPUArray.h"
//#include "/Users/lruthot/Downloads/cuda/include/hipDNN.h"
#include "hipDNN.h"
#include<time.h>
#include"convCuDNN2D.h"

/*
 * Host code
 * Y = convCouple(X,X_size,K,K_size,OP)
 OP : 0 for Conv, 1 for dYdK_T, 2 for dYdX_T
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare all variables.*/
    //char const * const 	errId = "parallel:gpu:mex:InvalidInput";
	void**				descriptorsAndMem;
    /* Initialize the MathWorks GPU API. */
	
	mxInitGPU();
	descriptorsAndMem = (void**)mxGetData(prhs[0]);
	cudaFreeAll(descriptorsAndMem);
}