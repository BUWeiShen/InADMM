#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
//#include "/Users/lruthot/Downloads/cuda/include/hipDNN.h"
#include "hipDNN.h"
#include "convCuDNN2D.h"
#include<time.h>

/*
 * Host code
 * Y = convCouple(X,X_size,K,K_size,OP)
 OP : 0 for Conv, 1 for dYdK_T, 2 for dYdX_T
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare all variables.*/
    mxGPUArray const	*gpuArrIn1;
    mxGPUArray const 	*gpuArrIn2;
	mxGPUArray 			*gpuArrOut = NULL;
    NUM_TYPE 	 		*ArrOut; // this is used for Y in Conv.
    NUM_TYPE const	 	*ArrIn1; // this is used for X in Conv.
	NUM_TYPE const 		*ArrIn2; // this is used as K in Conv.	
	
    int* 				X_size;
	int* 				K_size;
	int					stride;
	int 				ii;
    char const * const 	errId = "parallel:gpu:mex:InvalidInput";
	mwSize 				dims[4];
	const mwSize dimsBytes[] ={1};
	int 				Y_size_out[4];
	char* 				err = NULL;
	void*				descriptorsAndMemNew[NUM_ELEMENTS_SESSION];
	void**				descriptorsAndMem;
	size_t				nBytes;
	size_t*				pnBytes;
	
	Operation 			OP = Conv;
	
	int 				sessionExistsOutside;
    
	
	// clock_t ticks1, ticks2;

	
	
	
    /* Initialize the MathWorks GPU API. */
	// ticks1=clock();
	mxInitGPU();
	
    
    /* Throw an error if the input is not a GPU array. */
    if ((nrhs != 6)&&(nrhs!=7)) {
        mexErrMsgIdAndTxt(errId, "Number of arguments must be 6 or 7");
    }
	
	if (!(mxIsGPUArray(prhs[0]))||!(mxIsGPUArray(prhs[2]))) {
        mexErrMsgIdAndTxt(errId, "Input Kernel/Images are not GPU arrays");
    }
	
	sessionExistsOutside = nrhs == 7;
	
	X_size  	= (int*)mxGetData(prhs[4]);
	OP 			= (Operation)*X_size;
	
	X_size 		= (int*) mxGetData(prhs[1]);
    K_size  	= (int*) mxGetData(prhs[3]);
	
	stride      = *((int*) mxGetData(prhs[5]));
	
	
    
	 /*
     * Now that we have verified the data type, extract a pointer to the input
     * data on the device.
     */
	gpuArrIn1 = mxGPUCreateFromMxArray(prhs[0]);
	gpuArrIn2 = mxGPUCreateFromMxArray(prhs[2]);
	
	
    /*
     * Verify that A really is a single array before extracting the pointer.
     */
    if (mxGPUGetClassID(gpuArrIn1) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, "Arrays are not in single precision");
    }
	
	ArrIn1 	= (NUM_TYPE const *)(mxGPUGetDataReadOnly(gpuArrIn1));
	ArrIn2 	= (NUM_TYPE const *)(mxGPUGetDataReadOnly(gpuArrIn2));
	
	
	if (!sessionExistsOutside){
		descriptorsAndMem = descriptorsAndMemNew;
		for (ii = 0; ii < NUM_ELEMENTS_SESSION; ++ii){
			descriptorsAndMem[ii] = NULL;
		}
		err = createConvolutionDescriptors(descriptorsAndMem);
		if (err!=NULL){
			mxGPUDestroyGPUArray(gpuArrIn1);
			mxGPUDestroyGPUArray(gpuArrIn2);
			cudaFreeAll(descriptorsAndMem);
			mexErrMsgIdAndTxt(errId, err);
		}
		pnBytes 		  =	&nBytes;
	}else{
		descriptorsAndMem = (void**)mxGetData(prhs[6]);
		pnBytes			  = (size_t*) descriptorsAndMem[6];
		// mexPrintf("NUM Bytes is: %d\n",*pnBytes);
	}
	
	
	// OP == Conv: 		gpuArrIn1 = X ; gpuArrIn2 = K; gpuArrOut = Y;
	// OP == dYdK_T:	gpuArrIn1 = X ; gpuArrIn2 = dY;  gpuArrOut = dK;
	// OP == dYdX_T:	gpuArrIn1 = K ; gpuArrIn2 = dY;  gpuArrOut = dX;
	
	
	
	err = setConvolutionDescriptors(OP,X_size, K_size, Y_size_out,descriptorsAndMem,stride);
	if (err!=NULL){
		mxGPUDestroyGPUArray(gpuArrIn1);
		mxGPUDestroyGPUArray(gpuArrIn2);
		cudaFreeAll(descriptorsAndMem);
		mexErrMsgIdAndTxt(errId, err);
	}
	
	// ticks1=clock();
	if (OP==Conv){
		for (ii = 0; ii < 4; ++ii){
			dims[ii] = (mwSize)Y_size_out[ii];
		}
		// gpuArrOut = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(gpuArrIn1),dims,
		gpuArrOut = mxGPUCreateGPUArray(4,dims,
                            mxGPUGetClassID(gpuArrIn1),
                            mxGPUGetComplexity(gpuArrIn1),
                            MX_GPU_DO_NOT_INITIALIZE);
		ArrOut = (NUM_TYPE *)(mxGPUGetData(gpuArrOut));
	
		err = performConvolution(ArrIn1,ArrIn2,ArrOut,descriptorsAndMem,pnBytes);
		plhs[0] = mxGPUCreateMxArrayOnGPU(gpuArrOut);	
	
	}else if (OP == dYdK_T){
		for (ii = 0; ii < 4; ++ii){
			dims[ii] = (mwSize)K_size[ii];
		}
		// gpuArrOut = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(gpuArrIn1),dims,
		gpuArrOut = mxGPUCreateGPUArray(4,dims,
                            mxGPUGetClassID(gpuArrIn1),
                            mxGPUGetComplexity(gpuArrIn1),
                            MX_GPU_DO_NOT_INITIALIZE);
		ArrOut = (NUM_TYPE *)(mxGPUGetData(gpuArrOut));
	
		err = performConvolutiondYdK(ArrIn1,ArrIn2,ArrOut,descriptorsAndMem,pnBytes);
		plhs[0] = mxGPUCreateMxArrayOnGPU(gpuArrOut);
	}else if (OP == dYdX_T){
		for (ii = 0; ii < 4; ++ii){
			dims[ii] = (mwSize)X_size[ii];
		}
		// gpuArrOut = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(gpuArrIn2),dims,
		gpuArrOut = mxGPUCreateGPUArray(4,dims,
                            mxGPUGetClassID(gpuArrIn2),
                            mxGPUGetComplexity(gpuArrIn2),
                            MX_GPU_DO_NOT_INITIALIZE);
		ArrOut = (NUM_TYPE *)(mxGPUGetData(gpuArrOut));
	
		err = performConvolutiondYdX(ArrIn1,ArrIn2,ArrOut,descriptorsAndMem,pnBytes);
		plhs[0] = mxGPUCreateMxArrayOnGPU(gpuArrOut);
	}
	
    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */
	
    mxGPUDestroyGPUArray(gpuArrOut);
	mxGPUDestroyGPUArray(gpuArrIn1);
	mxGPUDestroyGPUArray(gpuArrIn2);
	// hipDeviceSynchronize();
	// ticks2=clock();
	// mexPrintf("Operation took: %lf, seconds\n",(double)(ticks2-ticks1)/(double)CLOCKS_PER_SEC);
	
	if (!sessionExistsOutside){
		cudaFreeAll(descriptorsAndMem);
	}else{
		hipDeviceSynchronize();
	}
	if (err != NULL){
		mexErrMsgIdAndTxt(errId, err);	
	}
}