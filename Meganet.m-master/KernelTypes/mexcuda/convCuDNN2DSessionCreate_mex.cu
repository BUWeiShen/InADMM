#include "mex.h"
#include "gpu/mxGPUArray.h"
//#include "/Users/lruthot/Downloads/cuda/include/hipDNN.h"
#include "hipDNN.h"
#include<time.h>
#include"convCuDNN2D.h"





/*
 * Host code
 * Y = convCouple(X,X_size,K,K_size,OP)
 OP : 0 for Conv, 1 for dYdK_T, 2 for dYdX_T
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare all variables.*/
 		

	int 				ii;
    char const * const 	errId = "parallel:gpu:mex:InvalidInput";
	char* 				err = NULL;
	void**				descriptorsAndMem;

	
	plhs[0] = mxCreateDoubleMatrix(NUM_ELEMENTS_SESSION, 1, mxREAL);
	
	
    /* Initialize the MathWorks GPU API. */
	mxInitGPU();
	
	descriptorsAndMem = (void**)mxGetData(plhs[0]);
	for (ii = 0; ii < NUM_ELEMENTS_SESSION; ++ii){
		descriptorsAndMem[ii] = NULL;
	}
	
	err = createConvolutionDescriptors(descriptorsAndMem);
	if (err!=NULL){
		cudaFreeAll(descriptorsAndMem);
		mexErrMsgIdAndTxt(errId, err);
	}
}