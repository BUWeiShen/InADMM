#include "hipDNN.h"
#include "mex.h"
#include "convCuDNN2D.h"

char* createConvolutionDescriptors(void** descriptorsAndMem){
	char* 							errMsg = NULL;
	size_t*							nBytes;
	hipdnnHandle_t 					hCudNN = NULL;
	hipdnnTensorDescriptor_t 		pXTensorDesc = NULL;
	hipdnnFilterDescriptor_t 		pKFilterDesc = NULL;
	hipdnnConvolutionDescriptor_t 	pConvDesc = NULL;
	hipdnnTensorDescriptor_t 		pYTensorDesc = NULL;
	hipdnnStatus_t 					status;

	status = hipdnnCreate(&hCudNN);
	descriptorsAndMem[0] = hCudNN;
	if (status != HIPDNN_STATUS_SUCCESS){
		mexPrintf("cudnn creat failed with error %d",status);
		errMsg = "hipdnnCreate failed";
		return errMsg;
	}

	status = hipdnnCreateTensorDescriptor(&pXTensorDesc);
	descriptorsAndMem[1] = pXTensorDesc;
	if (status != HIPDNN_STATUS_SUCCESS){
		errMsg = "hipdnnCreateTensorDescriptor for input failed";
		return errMsg;
	}

	status = hipdnnCreateFilterDescriptor(&pKFilterDesc);
	descriptorsAndMem[2] = pKFilterDesc;
	if (status != HIPDNN_STATUS_SUCCESS){
		errMsg = "hipdnnCreateFilterDescriptor failed";
		return errMsg;
	}

	status = hipdnnCreateConvolutionDescriptor(&pConvDesc);
	descriptorsAndMem[3] = pConvDesc;
	if (status != HIPDNN_STATUS_SUCCESS){
		errMsg = "hipdnnCreateConvolutionDescriptor failed";
		return errMsg;
	}
	

	status = hipdnnCreateTensorDescriptor(&pYTensorDesc);
	descriptorsAndMem[4] = pYTensorDesc;
	if (status != HIPDNN_STATUS_SUCCESS){
		errMsg = "hipdnnCreateTensorDescriptor for output failed";
		return errMsg;
	}
	nBytes = (size_t*)malloc(sizeof(size_t));
	// mexPrintf("nBytes allocated\n");
	*nBytes = 0;
	// mexPrintf("nBytes accessed\n");
	descriptorsAndMem[6] = (void*)nBytes;
	return NULL; // function successfully computed the size of the output.
}



char* setConvolutionDescriptors(Operation OP,int* IM_size,int* K_size,int* IM_size_out,void** descriptorsAndMem,int stride){
	char* 							errMsg = NULL;
	//hipdnnHandle_t 					hCudNN = NULL;
	hipdnnTensorDescriptor_t 		pXTensorDesc = NULL;
	hipdnnFilterDescriptor_t 		pKFilterDesc = NULL;
	hipdnnConvolutionDescriptor_t 	pConvDesc = NULL;
	hipdnnTensorDescriptor_t 		pYTensorDesc = NULL;
	hipdnnStatus_t 					status;
		
	// IM_size: We assume a NHWC format
	int n_in = IM_size[3]; // Number of images - originally 128
	int c_in = IM_size[2]; // Number of feature maps per image 
	int h_in = IM_size[1]; // Height of each image
	int w_in = IM_size[0]; // Width of each image  
	int cout_pFilter_in = K_size[3]; // Number of output feature maps  
	int cin_pFilter_in = c_in; // Number of input feature maps, should also equal K_size[2]
	int h_pFilter_in = K_size[1]; // Height of each pFilter
	int w_pFilter_in = K_size[0]; // Width of each pFilter
	int n_out = 0; // Number of output images.
	int c_out = 0; // Number of output feature maps per image.
	int h_out = 0; // Height of each output feature map.
	int w_out = 0; // Width of each output feature map.
	
	int h_pad = div(h_pFilter_in-1,2).quot;
	int w_pad = div(w_pFilter_in-1,2).quot; 

	/* to change to double, chance HIPDNN_DATA_FLOAT to HIPDNN_DATA_DOUBLE and change each float to double below */

	hipdnnDataType_t 				dataType 	  	= HIPDNN_DATA_FLOAT;
	hipdnnTensorFormat_t 			tensorFormat  	= HIPDNN_TENSOR_NCHW; //HIPDNN_TENSOR_NHWC HIPDNN_TENSOR_NCHW	
	hipdnnConvolutionMode_t      	convMode	  	= HIPDNN_CROSS_CORRELATION;// can also be HIPDNN_CONVOLUTION or HIPDNN_CROSS_CORRELATION

	if (K_size[2] != c_in){
		errMsg = "number of input channels in filter is different than the number of input channels in X";
		return errMsg;
	}
	
	// hCudNN 			= (hipdnnHandle_t)descriptorsAndMem[0];
	pXTensorDesc 	= (hipdnnTensorDescriptor_t)descriptorsAndMem[1];
	pKFilterDesc	= (hipdnnFilterDescriptor_t)descriptorsAndMem[2];
	pConvDesc		= (hipdnnConvolutionDescriptor_t)descriptorsAndMem[3];
	pYTensorDesc  	= (hipdnnTensorDescriptor_t)descriptorsAndMem[4];
	
	
//---------------------------------------
// Set (input) decriptors
//---------------------------------------
	status = hipdnnSetTensor4dDescriptor(pXTensorDesc, tensorFormat, dataType, n_in, c_in, h_in, w_in);
	if (status != HIPDNN_STATUS_SUCCESS){
		errMsg = "hipdnnSetTensor4dDescriptor failed";
		if (status == HIPDNN_STATUS_BAD_PARAM){
			mexPrintf("Images tensor parameters are (w,h,cin,N): %d,%d,%d,%d\n:",w_in,h_in,c_in,n_in);
			errMsg = "hipdnnSetTensor4dDescriptor failed with HIPDNN_STATUS_BAD_PARAM";
		}
		return errMsg;;
	}
	
	status = hipdnnSetFilter4dDescriptor(pKFilterDesc, dataType,tensorFormat, cout_pFilter_in, cin_pFilter_in, h_pFilter_in, w_pFilter_in);
	if (status != HIPDNN_STATUS_SUCCESS){
		errMsg = "hipdnnSetFilter4dDescriptor failed";
		if (status == HIPDNN_STATUS_BAD_PARAM){
			mexPrintf("Filter parameters are (w,h,cin,cout): %d,%d,%d,%d\n:",w_pFilter_in,h_pFilter_in,cin_pFilter_in,cout_pFilter_in);
			errMsg = "hipdnnSetFilter4dDescriptor failed with HIPDNN_STATUS_BAD_PARAM";
		}
		return errMsg;
	}

	// h_pad and w_pad assume "dirichlet 0 BC".
	status = hipdnnSetConvolution2dDescriptor(pConvDesc, h_pad, w_pad, stride, stride, 1, 1, convMode,dataType);
	if (status != HIPDNN_STATUS_SUCCESS){
		errMsg = "hipdnnSetConvolution2dDescriptor failed";
		return errMsg;
	}
	
//------------------------------------------------------------------------------
// Query output tensor, set output tensor descriptor, and allocate output layout
//------------------------------------------------------------------------------
	
	status = hipdnnGetConvolution2dForwardOutputDim(pConvDesc,pXTensorDesc,pKFilterDesc, &n_out, &c_out, &h_out, &w_out);
	if (status != HIPDNN_STATUS_SUCCESS){
		errMsg = "hipdnnGetConvolution2dForwardOutputDim failed";
		return errMsg;
	}
	IM_size_out[0] = w_out;IM_size_out[1] = h_out;IM_size_out[2] = c_out;IM_size_out[3] = n_out;
	
	status = hipdnnSetTensor4dDescriptor(pYTensorDesc, tensorFormat, dataType, IM_size_out[3], IM_size_out[2], IM_size_out[1], IM_size_out[0]);
	if (status != HIPDNN_STATUS_SUCCESS){
		errMsg = "hipdnnSetTensor4dDescriptor failed";
		return errMsg;
	}
	
	return NULL; // function successfully computed the size of the output.
}


char* updateAllocatedWorkspace(void** descriptorsAndMem, size_t* nBytesOld, size_t* nBytesNew){
	NUM_TYPE* 		pWorkspace	   = NULL;
	char* 	  		errMsg 		   = NULL;
	hipError_t 	err;
    if ((*nBytesNew > *nBytesOld) || (descriptorsAndMem[5]==NULL)){
		if (descriptorsAndMem[5]!=NULL) 
			hipFree(descriptorsAndMem[5]);
		err = hipMalloc((void**)&pWorkspace, *nBytesNew);
		descriptorsAndMem[5] = pWorkspace;
		*nBytesOld = *nBytesNew;
		if (err != hipSuccess){
			errMsg = "hipMalloc failed to allocate the workspace memory.";
			return errMsg;
		}
	}
	return NULL;
}


/*
	function performs a full NN forward convolution.
	input: 
		 - Y: is a pointer to populate the result. It is assumed to be already allocated with the right size (also see setup). 
	output:
		 - 	out param is an error massage string. If NULL, it is success. If not NULL - it contains a (hopefully informative) error 
			message that was raised by a cuDNN function.
*/

char* performConvolution(NUM_TYPE const *X, NUM_TYPE const *K, NUM_TYPE* Y, void** descriptorsAndMem, size_t* nBytesAllocated){
				
	char* 							errMsg = NULL;
	hipdnnHandle_t 					hCudNN = NULL;
	hipdnnTensorDescriptor_t 		pXTensorDesc = NULL;
	hipdnnFilterDescriptor_t 		pKFilterDesc = NULL;
	hipdnnConvolutionDescriptor_t 	pConvDesc = NULL;
	hipdnnTensorDescriptor_t 		pYTensorDesc = NULL;
	hipdnnStatus_t 					status;
	
	size_t							nBytesNeeded;
	/* to change to double, chance HIPDNN_DATA_FLOAT to HIPDNN_DATA_DOUBLE and change each float to double below */
	hipdnnConvolutionFwdAlgo_t 	convAlgo      = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;//HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
	
	NUM_TYPE*					pWorkspace	   = NULL;
	NUM_TYPE 					alpha 		   = 1.0;
	NUM_TYPE 					beta 		   = 0.0;

	hCudNN 			= (hipdnnHandle_t)descriptorsAndMem[0];
	pXTensorDesc 	= (hipdnnTensorDescriptor_t)descriptorsAndMem[1];
	pKFilterDesc	= (hipdnnFilterDescriptor_t)descriptorsAndMem[2];
	pConvDesc		= (hipdnnConvolutionDescriptor_t)descriptorsAndMem[3];
	pYTensorDesc  	= (hipdnnTensorDescriptor_t)descriptorsAndMem[4];
	
	
	status = hipdnnGetConvolutionForwardWorkspaceSize(hCudNN, pXTensorDesc, pKFilterDesc, pConvDesc, pYTensorDesc, convAlgo, &nBytesNeeded);
	if (status != HIPDNN_STATUS_SUCCESS){
		errMsg = "hipdnnGetConvolutionForwardWorkspaceSize failed";
		if (status == HIPDNN_STATUS_BAD_PARAM){
			errMsg = "hipdnnGetConvolutionForwardWorkspaceSize failed with HIPDNN_STATUS_BAD_PARAM";
		}
		if (status == HIPDNN_STATUS_NOT_SUPPORTED){
			errMsg = "hipdnnGetConvolutionForwardWorkspaceSize failed with HIPDNN_STATUS_NOT_SUPPORTED";
		}
		return errMsg;
	}
	
	errMsg = updateAllocatedWorkspace(descriptorsAndMem,nBytesAllocated,&nBytesNeeded);
	if (errMsg!= NULL)
		return errMsg;
	pWorkspace = (NUM_TYPE*)descriptorsAndMem[5];
//---------------------------------------
// Launch convolution on GPU
//---------------------------------------
 	status = hipdnnConvolutionForward(hCudNN, &alpha, pXTensorDesc, X, pKFilterDesc, K, pConvDesc, convAlgo, pWorkspace, *nBytesAllocated, &beta, pYTensorDesc, Y);
	if (status != HIPDNN_STATUS_SUCCESS){
		errMsg = "hipdnnConvolutionForward failed";
		if (status == HIPDNN_STATUS_BAD_PARAM){
			errMsg = "hipdnnConvolutionForward failed with HIPDNN_STATUS_BAD_PARAM";
		}
		return errMsg;
	}
	return NULL;
} 


char* performConvolutiondYdK(NUM_TYPE const *X, NUM_TYPE const *dY, NUM_TYPE* dK, void** descriptorsAndMem, size_t* nBytesAllocated){	
	char* 							errMsg = NULL;
	hipdnnHandle_t 					hCudNN = NULL;
	hipdnnTensorDescriptor_t 		pXTensorDesc = NULL;
	hipdnnFilterDescriptor_t 		pKFilterDesc = NULL;
	hipdnnConvolutionDescriptor_t 	pConvDesc = NULL;
	hipdnnTensorDescriptor_t 		pYTensorDesc = NULL;
	hipdnnStatus_t 					status;
	
	size_t							nBytesNeeded;

	/* to change to double, chance HIPDNN_DATA_FLOAT to HIPDNN_DATA_DOUBLE and change each float to double below */
	hipdnnConvolutionBwdFilterAlgo_t dYdKBwdAlgo 	= HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1;
	
	NUM_TYPE*					pWorkspace	   = NULL;
	NUM_TYPE 					alpha 		   = 1.0;
	NUM_TYPE 					beta 		   = 0.0;
	

	hCudNN 			= (hipdnnHandle_t)descriptorsAndMem[0];
	pXTensorDesc 	= (hipdnnTensorDescriptor_t)descriptorsAndMem[1];
	pKFilterDesc	= (hipdnnFilterDescriptor_t)descriptorsAndMem[2];
	pConvDesc		= (hipdnnConvolutionDescriptor_t)descriptorsAndMem[3];
	pYTensorDesc  	= (hipdnnTensorDescriptor_t)descriptorsAndMem[4];
	pWorkspace		= (NUM_TYPE*)descriptorsAndMem[5];
	
	status = hipdnnGetConvolutionBackwardFilterWorkspaceSize(hCudNN,pXTensorDesc,pYTensorDesc,pConvDesc,pKFilterDesc,dYdKBwdAlgo,&nBytesNeeded);
	if (status != HIPDNN_STATUS_SUCCESS){
		errMsg = "hipdnnGetConvolutionBackwardFilterWorkspaceSize failed";
		if (status == HIPDNN_STATUS_BAD_PARAM){
			errMsg = "hipdnnGetConvolutionBackwardFilterWorkspaceSize failed with HIPDNN_STATUS_BAD_PARAM";
		}
		if (status == HIPDNN_STATUS_NOT_SUPPORTED){
			errMsg = "hipdnnGetConvolutionBackwardFilterWorkspaceSize failed with HIPDNN_STATUS_NOT_SUPPORTED";
		}
		return errMsg;
	}
	errMsg = updateAllocatedWorkspace(descriptorsAndMem,nBytesAllocated,&nBytesNeeded);
	if (errMsg!= NULL)
		return errMsg;
	
	pWorkspace = (NUM_TYPE*)descriptorsAndMem[5];
	
//---------------------------------------
// Launch convolution on GPU
//---------------------------------------
 	status = hipdnnConvolutionBackwardFilter(hCudNN, &alpha, pXTensorDesc, X, pYTensorDesc, dY, pConvDesc, dYdKBwdAlgo, pWorkspace, *nBytesAllocated, &beta, pKFilterDesc, dK);
	if (status != HIPDNN_STATUS_SUCCESS){
		errMsg = "hipdnnConvolutionBackwardFilter failed";
		if (status == HIPDNN_STATUS_BAD_PARAM){
			errMsg = "hipdnnConvolutionBackwardFilter failed with HIPDNN_STATUS_BAD_PARAM";
		}
		return errMsg;
	}
	return NULL;
} 


char* performConvolutiondYdX(NUM_TYPE const *K, NUM_TYPE const *dY, NUM_TYPE* dX, void** descriptorsAndMem, size_t* nBytesAllocated){	
	char* 							errMsg = NULL;
	hipdnnHandle_t 					hCudNN = NULL;
	hipdnnTensorDescriptor_t 		pXTensorDesc = NULL;
	hipdnnFilterDescriptor_t 		pKFilterDesc = NULL;
	hipdnnConvolutionDescriptor_t 	pConvDesc = NULL;
	hipdnnTensorDescriptor_t 		pYTensorDesc = NULL;
	hipdnnStatus_t 					status;
	
	size_t							nBytesNeeded;


	/* to change to double, chance HIPDNN_DATA_FLOAT to HIPDNN_DATA_DOUBLE and change each float to double below */
	hipdnnConvolutionBwdDataAlgo_t dYdXBwdAlgo 	= HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0;
	
	NUM_TYPE*					pWorkspace	   = NULL;
	NUM_TYPE 					alpha 		   = 1.0;
	NUM_TYPE 					beta 		   = 0.0;

	hCudNN 			= (hipdnnHandle_t)descriptorsAndMem[0];
	pXTensorDesc 	= (hipdnnTensorDescriptor_t)descriptorsAndMem[1];
	pKFilterDesc	= (hipdnnFilterDescriptor_t)descriptorsAndMem[2];
	pConvDesc		= (hipdnnConvolutionDescriptor_t)descriptorsAndMem[3];
	pYTensorDesc  	= (hipdnnTensorDescriptor_t)descriptorsAndMem[4];
	pWorkspace		= (NUM_TYPE*)descriptorsAndMem[5];
	
	status = hipdnnGetConvolutionBackwardDataWorkspaceSize(hCudNN,pKFilterDesc,pYTensorDesc,pConvDesc,pXTensorDesc,dYdXBwdAlgo,&nBytesNeeded);
	if (status != HIPDNN_STATUS_SUCCESS){
		errMsg = "hipdnnGetConvolutionBackwardDataWorkspaceSize failed";
		if (status == HIPDNN_STATUS_BAD_PARAM){
			errMsg = "hipdnnGetConvolutionBackwardDataWorkspaceSize failed with HIPDNN_STATUS_BAD_PARAM";
		}
		if (status == HIPDNN_STATUS_NOT_SUPPORTED){
			errMsg = "hipdnnGetConvolutionBackwardDataWorkspaceSize failed with HIPDNN_STATUS_NOT_SUPPORTED";
		}
		return errMsg;
	}
	
	errMsg = updateAllocatedWorkspace(descriptorsAndMem,nBytesAllocated,&nBytesNeeded);
	pWorkspace = (NUM_TYPE*)descriptorsAndMem[5];
	if (errMsg!= NULL)
		return errMsg;
//---------------------------------------
// Launch convolution on GPU
//---------------------------------------
	
 	status = hipdnnConvolutionBackwardData(hCudNN, &alpha, pKFilterDesc, K, pYTensorDesc, dY, pConvDesc, dYdXBwdAlgo, pWorkspace, *nBytesAllocated, &beta, pXTensorDesc, dX);
	if (status != HIPDNN_STATUS_SUCCESS){
		errMsg = "hipdnnConvolutionBackwardData failed";
		if (status == HIPDNN_STATUS_BAD_PARAM){
			errMsg = "hipdnnConvolutionBackwardData failed with HIPDNN_STATUS_BAD_PARAM";
		}
		return errMsg;
	}
	return NULL;
} 
 

void cudaFreeAll(void** descriptorsAndMem){
	int ii;
	if (descriptorsAndMem[0] != NULL)
		hipdnnDestroy((hipdnnHandle_t)descriptorsAndMem[0]);// hCudNN
	
	if (descriptorsAndMem[1] != NULL)
		hipdnnDestroyTensorDescriptor((hipdnnTensorDescriptor_t)descriptorsAndMem[1]); //pXTensorDesc

	if (descriptorsAndMem[2] != NULL)
		hipdnnDestroyFilterDescriptor((hipdnnFilterDescriptor_t)descriptorsAndMem[2]);//pKFilterDesc

	if (descriptorsAndMem[3] != NULL)
		hipdnnDestroyConvolutionDescriptor((hipdnnConvolutionDescriptor_t)descriptorsAndMem[3]);//pConvDesc

	if (descriptorsAndMem[4] != NULL)
		hipdnnDestroyTensorDescriptor((hipdnnTensorDescriptor_t)descriptorsAndMem[4]);//pYTensorDesc

	if (descriptorsAndMem[5] != NULL)
		hipFree(descriptorsAndMem[5]); //pWorkspace
	
	if (descriptorsAndMem[6] != NULL)
		free(descriptorsAndMem[6]); //nBytes
	
	for (ii = 0; ii < NUM_ELEMENTS_SESSION; ++ii){
		descriptorsAndMem[ii] = NULL;
	}
}